#include "hip/hip_runtime.h"
#include <vector> 
#include <iostream> 
#include <thrust/transform.h> 
#include <thrust/functional.h> 
#include <thrust/host_vector.h> 
#include <thrust/device_vector.h> 

using namespace std;
using namespace thrust;

__global__ void rasterizeBlock(uchar4 *frags, int *w, int *h) {
	for (int y = 0; y < *h; y++)
		for (int x = 0; x < *w; x++)
			frags[x + y * (*w)] = make_uchar4(255, 0, 0, 255);
}

uchar4* rasterize(int w, int h, float3 *vertices, int *indices) {
	uchar4* frags = (uchar4 *)malloc(sizeof(uchar4) * w * h);;

	float3 v1 = *(vertices + indices[0]),
		v2 = *(vertices + indices[1]),
		v3 = *(vertices + indices[2]);

	float x1 = fmin(v1.x, fmin(v2.x, v3.x)) * w,
		x2 = fmax(v1.x, fmax(v2.x, v3.x)) * w,
		y1 = fmin(v1.y, fmin(v2.y, v3.y)) * h,
		y2 = fmax(v1.y, fmax(v2.y, v3.y)) * h;

	printf("%f, %f, %f, %f", x1, x2, y1, y2);

	int bX = x1, bY = y1, bW = x2 - x1, bH = y2 - y1;
	int *d_bW, *d_bH;
	uchar4* block = (uchar4 *)malloc(sizeof(uchar4) * bW * bH);
	uchar4* d_block;

	hipMalloc((void **) &d_bW, sizeof(int));
	hipMalloc((void **) &d_bH, sizeof(int));
	hipMalloc((void **) &d_block, sizeof(uchar4) * bW * bH);

	hipMemcpy(d_bW, &bW, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_bH, &bH, sizeof(int), hipMemcpyHostToDevice);

	rasterizeBlock<<<1,1>>>(d_block, d_bW, d_bH);
	
	hipMemcpy(block, d_block, sizeof(uchar4) * bW * bH, hipMemcpyDeviceToHost);

	for (int y = 0; y < bH; y++)
		for (int x = 0; x < bW; x++)
			frags[x + bX + w * (y + bY)] = block[x + y * bW];

	free(block);
	hipFree(d_bW);
	hipFree(d_bH);
	hipFree(d_block);

	return frags;
}

extern "C" int test(void) {
	vector<float>  vh = { 0, 1, 2, 3, 4, 5, 6, 7 };
	device_vector<float> v = vh;
	device_vector<float> v_out(v.size());
	thrust::transform(v.begin(), v.end(), v_out.begin(),
		[=] __device__(float x) {
		return x*42 + 7;
	});
	for (size_t i = 0; i < v_out.size(); i++)
		std::cout << v_out[i] << std::endl;
	return 0;
}