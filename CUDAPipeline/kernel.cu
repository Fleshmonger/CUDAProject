#include "hip/hip_runtime.h"
#include <vector> 
#include <iostream> 
#include <thrust/transform.h> 
#include <thrust/functional.h> 
#include <thrust/host_vector.h> 
#include <thrust/device_vector.h> 

using namespace std;
using namespace thrust;

__global__ void rasterizeTriangle(uchar4 *pixels, int *width, int *height, float3 *vertices, int3 *indices) {
	int3 index = indices[blockIdx.x];
	float3 v1 = vertices[index.x],
		v2 = vertices[index.y],
		v3 = vertices[index.z];
	int left = fmin(v1.x, fmin(v2.x, v3.x)) * (*width),
		right = fmax(v1.x, fmax(v2.x, v3.x)) * (*width),
		bottom = fmin(v1.y, fmin(v2.y, v3.y)) * (*height),
		top = fmax(v1.y, fmax(v2.y, v3.y)) * (*height);
	for (int x = left; x < right; x++)
		for (int y = bottom; y < top; y++)
			pixels[x + y * (*width)] = make_uchar4(255, 0, 0, 255);

}

void rasterize(uchar4 *pixels, int width, int height, float3 *vertices, int3 *indices, int vLength, int iLength) {
	uchar4 *d_pixels;
	int *d_width, *d_height;
	float3 *d_vertices;
	int3 *d_indices;

	hipMalloc((void **)&d_pixels, sizeof(uchar4) * width * height);
	hipMalloc((void **)&d_width, sizeof(int));
	hipMalloc((void **)&d_height, sizeof(int));
	hipMalloc((void **)&d_vertices, sizeof(float3) * vLength);
	hipMalloc((void **)&d_indices, sizeof(int3) * iLength);

	hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_height, &height, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_vertices, vertices, sizeof(float3) * vLength, hipMemcpyHostToDevice);
	hipMemcpy(d_indices, indices, sizeof(int3) * iLength, hipMemcpyHostToDevice);

	rasterizeTriangle<<<1, 1 >>>(d_pixels, d_width, d_height, d_vertices, d_indices);

	hipMemcpy(pixels, d_pixels, sizeof(uchar4) * width * height, hipMemcpyDeviceToHost);

	hipFree(d_pixels);
	hipFree(d_width);
	hipFree(d_height);
	hipFree(d_vertices);
	hipFree(d_indices);
}