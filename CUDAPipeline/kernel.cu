#include "hip/hip_runtime.h"
#include <vector> 
#include <iostream> 
#include <thrust/transform.h> 
#include <thrust/functional.h> 
#include <thrust/host_vector.h> 
#include <thrust/device_vector.h> 

using namespace std;
using namespace thrust;

#define THREADS_PER_BLOCK 64
#define SQRT_TPB 8

struct fragment {
	bool *pixels;
	int i_x, i_y, i_width, i_height, index;

	__device__  fragment() {
		this->pixels = nullptr;
		this->i_x = 0;
		this->i_y = 0;
		this->i_width = 0;
		this->i_height = 0;
		this->index = 0;
	}

	__device__ fragment(bool *pixels, int i_x, int i_y, int i_width, int i_height, int index) {
		this->pixels = pixels;
		this->i_x = i_x;
		this->i_y = i_y;
		this->i_width = i_width;
		this->i_height = i_height;
		this->index = index;
	}
};

// Runs the vertex shader on a vertex.
__global__ void vertexShader(int *width, int *height, float3 *vertices) {

}


// Rasterizes a triangle.
__global__ void rasterizeTriangle(int *width, int *height, float3 *vertices, int3 *indices, fragment *fragments) {
	// Retrieve Vertices
	int3 index = indices[blockIdx.x];
	float3 v1 = vertices[index.x],
		v2 = vertices[index.y],
		v3 = vertices[index.z];

	// Image Coordinates
	float i_v1x = (v1.x / 2.0 + 0.5) * (*width),
		i_v1y = (v1.y / 2.0 + 0.5) * (*height),
		i_v2x = (v2.x / 2.0 + 0.5) * (*width),
		i_v2y = (v2.y / 2.0 + 0.5) * (*height),
		i_v3x = (v3.x / 2.0 + 0.5) * (*width),
		i_v3y = (v3.y / 2.0 + 0.5) * (*height);

	// Triangle Bounding Box
	float t_left = fmin(v1.x, fmin(v2.x, v3.x)) * (*width),
		t_right = fmax(v1.x, fmax(v2.x, v3.x)) * (*width),
		t_bottom = fmin(v1.y, fmin(v2.y, v3.y)) * (*height),
		t_top = fmax(v1.y, fmax(v2.y, v3.y)) * (*height);

	// Fragment Dimensions
	int f_width = ceil((t_right - t_left) / SQRT_TPB),
		f_height = ceil((t_top - t_bottom) / SQRT_TPB),
		f_x = t_left + (threadIdx.x % SQRT_TPB) * f_width,
		f_y = t_bottom + (threadIdx.x / SQRT_TPB) * f_height;

	// Barycentric Init
	float alpha_denom = (i_v2y - i_v3y) * (i_v1x - i_v3x) + (i_v3x - i_v2x) * (i_v1y - i_v3y),
		beta_denom = (i_v2y - i_v3y) * (i_v1x - i_v3x) + (i_v3x - i_v2x) * (i_v1y - i_v3y);

	// Init Pixels
	bool *f_pixels = (bool*)malloc(sizeof(bool) * f_width * f_height);
	bool f_empty = true;

	// Rasterize
	for (int x = 0; x < f_width; x++) {
		for (int y = 0; y < f_height; y++) {
			float i_x = f_x + x + 0.5, i_y = f_y + y + 0.5,
				alpha = ((i_v2y - i_v3y) * (i_x - i_v3x) + (i_v3x - i_v2x) * (i_y - i_v3y)) / alpha_denom,
				beta = ((i_v3y - i_v1y) * (i_x - i_v3x) + (i_v1x - i_v3x) * (i_y - i_v3y)) / beta_denom,
				gamma = 1.0f - alpha - beta;
			if (0.0 <= alpha && 0.0 <= beta && 0.0 <= gamma) {
				f_pixels[x + y * f_width] = true;
				f_empty = false;
			}
			else
				f_pixels[x + y * f_width] = false;
		}
	}
	if (f_empty)
		free(f_pixels);
	else
		fragments[threadIdx.x + blockIdx.x * THREADS_PER_BLOCK] = fragment(f_pixels, f_x, f_y, f_width, f_height, blockIdx.x);
}

// Runs the fragment shader on a fragment.
__global__ void fragmentShader(uchar4 *d_pixels, int *width, float3 *vertices, int3 *indices, fragment *fragments) {
	fragment frag = fragments[threadIdx.x + blockIdx.x * THREADS_PER_BLOCK];
	if (frag.pixels == nullptr)
		return;
	for (int x = 0; x < frag.i_width; x++) {
		for (int y = 0; y < frag.i_height; y++) {
			if (frag.pixels[x + y * frag.i_width])
				d_pixels[frag.i_x + x + (frag.i_y + y) * (*width)] = make_uchar4(255, 0, 0, 255);
		}
	}
	free(frag.pixels);
}

void pipeline(uchar4 *d_pixels, int *d_width, int *d_height, float3 *d_vertices, int3 *d_indices, int numVertices, int numTriangles, fragment *d_fragments) {
	// Vertex Shader
	vertexShader<<<numVertices / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_width, d_height, d_vertices);

	// Rasterize
	rasterizeTriangle<<<numTriangles, THREADS_PER_BLOCK>>>(d_width, d_height, d_vertices, d_indices, d_fragments);

	// Fragment Shader
	fragmentShader<<<numTriangles, THREADS_PER_BLOCK>>>(d_pixels, d_width, d_vertices, d_indices, d_fragments);

	// Geometry Shader
}

void draw(uchar4 *pixels, int width, int height, float3 *vertices, int3 *indices, int numVertices, int numTriangles) {
	uchar4 *d_pixels;
	int *d_width, *d_height;
	float3 *d_vertices;
	int3 *d_indices;
	fragment *d_fragments;

	hipMalloc((void **)&d_pixels, sizeof(uchar4) * width * height);
	hipMalloc((void **)&d_width, sizeof(int));
	hipMalloc((void **)&d_height, sizeof(int));
	hipMalloc((void **)&d_vertices, sizeof(float3) * numVertices);
	hipMalloc((void **)&d_indices, sizeof(int3) * numTriangles);
	hipMalloc((void **)&d_fragments, sizeof(fragment) * numTriangles * THREADS_PER_BLOCK);

	hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_height, &height, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_vertices, vertices, sizeof(float3) * numVertices, hipMemcpyHostToDevice);
	hipMemcpy(d_indices, indices, sizeof(int3) * numTriangles, hipMemcpyHostToDevice);

	pipeline(d_pixels, d_width, d_height, d_vertices, d_indices, numVertices, numTriangles, d_fragments);

	hipMemcpy(pixels, d_pixels, sizeof(uchar4) * width * height, hipMemcpyDeviceToHost);

	hipFree(d_pixels);
	hipFree(d_width);
	hipFree(d_height);
	hipFree(d_vertices);
	hipFree(d_indices);
	hipFree(d_fragments);
}